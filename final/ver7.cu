#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>
#include <hip/hip_runtime.h>

#include <hip/hip_complex.h>
#include <time.h>

#define M_PI 3.14159265358979323846

using namespace std;

typedef vector<vector<hipDoubleComplex>> MatrixD;

// Helper functions for cuDoubleComplex operations
__device__ inline hipDoubleComplex multiplyComplex(hipDoubleComplex a, hipDoubleComplex b) {
    return make_hipDoubleComplex(
        a.x * b.x - a.y * b.y,
        a.x * b.y + a.y * b.x
    );
}

__device__ inline hipDoubleComplex addComplex(hipDoubleComplex a, hipDoubleComplex b) {
    return make_hipDoubleComplex(a.x + b.x, a.y + b.y);
}

__device__ inline hipDoubleComplex subtractComplex(hipDoubleComplex a, hipDoubleComplex b) {
    return make_hipDoubleComplex(a.x - b.x, a.y - b.y);
}

__device__ inline double absComplex(hipDoubleComplex a) {
    return sqrt(a.x * a.x + a.y * a.y);
}

// Previous utility functions remain the same
inline int power_of_two(int n) {
    int i = 1;
    while (i < n)
        i <<= 1;
    return i;
}

inline int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

inline void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_GRAY, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

void generate_bit_reversal_indices(int* bit_reversed_indices, int n) {
    int log2_n = log2(n);  // 假設 n 是 2 的冪次
    for (int i = 0; i < n; i++) {
        int reversed = 0;
        int value = i;
        for (int j = 0; j < log2_n; j++) {
            reversed = (reversed << 1) | (value & 1);  // 將每一位翻轉
            value >>= 1;  // 右移
        }
        bit_reversed_indices[i] = reversed;
    }
}

__global__ void bit_reversal_reorder_kernel(
    hipDoubleComplex* input, hipDoubleComplex* output, int* bit_reversed_indices, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int reversed_idx = bit_reversed_indices[idx];
        output[reversed_idx] = input[idx];
    }
}

// CUDA kernel for frequency centering
__global__ void center_frequency_kernel(hipDoubleComplex* data, int height, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int half_h = height / 2;
    int half_w = width / 2;
    
    if (idx < half_w && idy < half_h) {
        int pos1 = idy * width + idx;
        int pos2 = (idy + half_h) * width + (idx + half_w);
        int pos3 = idy * width + (idx + half_w);
        int pos4 = (idy + half_h) * width + idx;
        
        hipDoubleComplex temp1 = data[pos1];
        hipDoubleComplex temp2 = data[pos3];
        
        data[pos1] = data[pos2];
        data[pos2] = temp1;
        data[pos3] = data[pos4];
        data[pos4] = temp2;
    }
}

__global__ void fft_kernel_row(hipDoubleComplex* data, int n, int stage, int row, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int m = 1 << stage;
    int m2 = m >> 1;

    if (idx < n / 2) {
        int group = idx / m2;
        int pair = idx % m2;
        int pos = row * width + group * m + pair;

        double angle = -2.0 * M_PI * pair / m;
        hipDoubleComplex w = make_hipDoubleComplex(cos(angle), sin(angle));
        hipDoubleComplex t = multiplyComplex(w, data[pos + m2]);
        hipDoubleComplex u = data[pos];

        data[pos] = addComplex(u, t);
        data[pos + m2] = subtractComplex(u, t);
    }
}

__global__ void fft_kernel_col(hipDoubleComplex* data, int n, int stage, int col, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int m = 1 << stage;
    int m2 = m >> 1;

    if (idx < n / 2) {
        int group = idx / m2;
        int pair = idx % m2;
        int pos = (group * m + pair) * width + col;

        double angle = -2.0 * M_PI * pair / m;
        hipDoubleComplex w = make_hipDoubleComplex(cos(angle), sin(angle));
        hipDoubleComplex t = multiplyComplex(w, data[pos + m2 * width]);
        hipDoubleComplex u = data[pos];

        data[pos] = addComplex(u, t);
        data[pos + m2 * width] = subtractComplex(u, t);
    }
}

__global__ void bit_reversal_col_kernel(hipDoubleComplex* data, int* indices, int n, int col, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int reversed_idx = indices[idx];
        if (idx < reversed_idx) {
            hipDoubleComplex temp = data[idx * width + col];
            data[idx * width + col] = data[reversed_idx * width + col];
            data[reversed_idx * width + col] = temp;
        }
    }
}

// 用於計算最大值的 kernel
__global__ void compute_max_val_kernel(hipDoubleComplex* data, double* max_val, int height, int width, int w) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < width && idy < height) {
        double val = log(1 + absComplex(data[idy * w + idx]));
        atomicMax((unsigned long long*)max_val, __double_as_longlong(val));
    }
}

// 用於將複數數據轉換為圖像數據的 kernel
__global__ void convert_to_image_kernel(hipDoubleComplex* data, unsigned char* output, 
                                      double* max_val, int height, int width, int w) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < width && idy < height) {
        double val = log(1 + absComplex(data[idy * w + idx]));
        output[idy * width + idx] = static_cast<unsigned char>((val / *max_val) * 255);
    }
}

// CUDA kernel for grayscale conversion
__global__ void convert_to_gray_kernel(unsigned char *s, unsigned char *t, int height, int width, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < width && idy < height) {
        int pos = idy * width + idx;
        int pos_rgb = idy * width * channels + idx * channels;
        
        int R = s[pos_rgb + 2];
        int G = s[pos_rgb + 1];
        int B = s[pos_rgb + 0];
        
        t[pos] = 0.299f * R + 0.587f * G + 0.114f * B;
    }
}

// Move kernel to global scope
__global__ void init_fft_data(unsigned char* gray, hipDoubleComplex* fft_data, 
                             int height, int width, int w, int h) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < w && idy < h) {
        if (idx < width && idy < height) {
            fft_data[idy * w + idx] = make_hipDoubleComplex((double)gray[idy * width + idx], 0);
        } else {
            fft_data[idy * w + idx] = make_hipDoubleComplex(0, 0);
        }
    }
}

// Modified main processing function that keeps data on GPU
void process_image_gpu(unsigned char* h_input, unsigned char* h_output, 
                      int height, int width, int channels) {
    unsigned char *d_input, *d_gray;
    hipDoubleComplex *d_fft_data;
    unsigned char *d_output;
    double *d_max_val;
    int *d_indices;
    
    size_t input_size = height * width * channels * sizeof(unsigned char);
    size_t gray_size = height * width * sizeof(unsigned char);
    int w = power_of_two(width);
    int h = power_of_two(height);
    size_t complex_size = h * w * sizeof(hipDoubleComplex);
    
    hipMalloc(&d_input, input_size);
    hipMalloc(&d_gray, gray_size);
    hipMalloc(&d_fft_data, complex_size);
    hipMalloc(&d_output, gray_size);
    hipMalloc(&d_max_val, sizeof(double));
    hipMalloc(&d_indices, max(w, h) * sizeof(int));
    
    hipMemcpy(d_input, h_input, input_size, hipMemcpyHostToDevice);
    
    dim3 blockDim(32, 32);
    dim3 gridDim(
        (width + blockDim.x - 1) / blockDim.x,
        (height + blockDim.y - 1) / blockDim.y
    );
    
    convert_to_gray_kernel<<<gridDim, blockDim>>>(d_input, d_gray, height, width, channels);
    hipMemset(d_max_val, 0, sizeof(double));
    init_fft_data<<<gridDim, blockDim>>>(d_gray, d_fft_data, height, width, w, h);
    
    int threadsPerBlock = 256;
    int numBlocks;
    
    int* h_indices = (int*)malloc(max(w, h) * sizeof(int));
    generate_bit_reversal_indices(h_indices, w);
    hipMemcpy(d_indices, h_indices, w * sizeof(int), hipMemcpyHostToDevice);
    
    for (int i = 0; i < h; ++i) {
        hipDoubleComplex* row = d_fft_data + i * w;
        numBlocks = (w + threadsPerBlock - 1) / threadsPerBlock;
        
        bit_reversal_reorder_kernel<<<numBlocks, threadsPerBlock>>>(
            row, row, d_indices, w);
        
        for (int stage = 1; stage <= (int)log2(w); ++stage) {
            fft_kernel_row<<<numBlocks, threadsPerBlock>>>(d_fft_data, w, stage, i, w);
        }
    }
    
    generate_bit_reversal_indices(h_indices, h);
    hipMemcpy(d_indices, h_indices, h * sizeof(int), hipMemcpyHostToDevice);
    
    for (int j = 0; j < w; ++j) {
        numBlocks = (h + threadsPerBlock - 1) / threadsPerBlock;
        
        bit_reversal_col_kernel<<<numBlocks, threadsPerBlock>>>(
            d_fft_data, d_indices, h, j, w);
        
        for (int stage = 1; stage <= (int)log2(h); ++stage) {
            fft_kernel_col<<<(h/2 + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(
                d_fft_data, h, stage, j, w);
        }
    }
    
    center_frequency_kernel<<<gridDim, blockDim>>>(d_fft_data, h, w);
    compute_max_val_kernel<<<gridDim, blockDim>>>(d_fft_data, d_max_val, height, width, w);
    convert_to_image_kernel<<<gridDim, blockDim>>>(d_fft_data, d_output, d_max_val, 
                                                  height, width, w);
    
    hipMemcpy(h_output, d_output, gray_size, hipMemcpyDeviceToHost);
    
    hipFree(d_input);
    hipFree(d_gray);
    hipFree(d_fft_data);
    hipFree(d_output);
    hipFree(d_max_val);
    hipFree(d_indices);
    free(h_indices);
}

// Modified main function
int main(int argc, char **argv) {
    struct timespec start, end;
    double time_used;
    
    // 開始計時
    clock_gettime(CLOCK_MONOTONIC, &start);

    unsigned height, width, channels;
    unsigned char *src = NULL, *dst;

    // Read input
    read_png(argv[1], &src, &height, &width, &channels);
    dst = (unsigned char *) malloc(height * width * sizeof(unsigned char));

    // Process everything on GPU
    process_image_gpu(src, dst, height, width, channels);

    // Write output
    write_png(argv[2], dst, height, width, 1);

    // Clean up
    free(src);
    free(dst);

    // 結束計時
    clock_gettime(CLOCK_MONOTONIC, &end);
    
    // 計算執行時間（轉換為毫秒）
    time_used = (end.tv_sec - start.tv_sec) * 1000.0 + 
                (end.tv_nsec - start.tv_nsec) / 1000000.0;
    
    printf("Total execution time: %.2f ms\n", time_used);
    return 0;
}
// 複數型別改成 cuDoubleComplex