#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <cassert>
#include <zlib.h>
#include <png.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <time.h>

#define M_PI 3.14159265358979323846

using namespace std;

typedef complex<double> ComplexD;
typedef vector<vector<ComplexD>> MatrixD;
typedef thrust::complex<double> ComplexGPU;

inline int power_of_two(int n) {
    int i = 1;
    while (i < n)
        i <<= 1;
    return i;
}

inline int read_png(const char* filename, unsigned char** image, unsigned* height, unsigned* width, unsigned* channels) {

    unsigned char sig[8];
    FILE* infile;
    infile = fopen(filename, "rb");

    fread(sig, 1, 8, infile);
    if (!png_check_sig(sig, 8))
        return 1;   /* bad signature */

    png_structp png_ptr;
    png_infop info_ptr;

    png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png_ptr)
        return 4;   /* out of memory */
  
    info_ptr = png_create_info_struct(png_ptr);
    if (!info_ptr) {
        png_destroy_read_struct(&png_ptr, NULL, NULL);
        return 4;   /* out of memory */
    }

    png_init_io(png_ptr, infile);
    png_set_sig_bytes(png_ptr, 8);
    png_read_info(png_ptr, info_ptr);
    int bit_depth, color_type;
    png_get_IHDR(png_ptr, info_ptr, width, height, &bit_depth, &color_type, NULL, NULL, NULL);

    png_uint_32  i, rowbytes;
    png_bytep  row_pointers[*height];
    png_read_update_info(png_ptr, info_ptr);
    rowbytes = png_get_rowbytes(png_ptr, info_ptr);
    *channels = (int) png_get_channels(png_ptr, info_ptr);

    if ((*image = (unsigned char *) malloc(rowbytes * *height)) == NULL) {
        png_destroy_read_struct(&png_ptr, &info_ptr, NULL);
        return 3;
    }

    for (i = 0;  i < *height;  ++i)
        row_pointers[i] = *image + i * rowbytes;
    png_read_image(png_ptr, row_pointers);
    png_read_end(png_ptr, NULL);
    return 0;
}

inline void write_png(const char* filename, png_bytep image, const unsigned height, const unsigned width, const unsigned channels) {
    FILE* fp = fopen(filename, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_set_IHDR(png_ptr, info_ptr, width, height, 8,
                 PNG_COLOR_TYPE_GRAY, PNG_INTERLACE_NONE,
                 PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT);
    png_set_filter(png_ptr, 0, PNG_NO_FILTERS);
    png_write_info(png_ptr, info_ptr);
    png_set_compression_level(png_ptr, 1);

    png_bytep row_ptr[height];
    for (int i = 0; i < height; ++ i) {
        row_ptr[i] = image + i * width * channels * sizeof(unsigned char);
    }
    png_write_image(png_ptr, row_ptr);
    png_write_end(png_ptr, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    fclose(fp);
}

void generate_bit_reversal_indices(int* bit_reversed_indices, int n) {
    int log2_n = log2(n);  // 假設 n 是 2 的冪次
    for (int i = 0; i < n; i++) {
        int reversed = 0;
        int value = i;
        for (int j = 0; j < log2_n; j++) {
            reversed = (reversed << 1) | (value & 1);  // 將每一位翻轉
            value >>= 1;  // 右移
        }
        bit_reversed_indices[i] = reversed;
    }
}

__global__ void bit_reversal_reorder_kernel(
    thrust::complex<double>* input, thrust::complex<double>* output, int* bit_reversed_indices, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int reversed_idx = bit_reversed_indices[idx];
        output[reversed_idx] = input[idx];  // 根據反轉後的索引重排數據
    }
}

void gpu_bit_reversal(thrust::complex<double>* data, int n) {
    // 1. 在 CPU 上生成位元反轉索引表
    int* bit_reversed_indices = (int*)malloc(n * sizeof(int));
    generate_bit_reversal_indices(bit_reversed_indices, n);

    // 2. 在 GPU 上分配記憶體
    thrust::complex<double> *d_input, *d_output;
    int* d_indices;

    hipMalloc(&d_input, n * sizeof(thrust::complex<double>));
    hipMalloc(&d_output, n * sizeof(thrust::complex<double>));
    hipMalloc(&d_indices, n * sizeof(int));

    // 3. 拷貝數據和索引表到 GPU
    hipMemcpy(d_input, data, n * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, bit_reversed_indices, n * sizeof(int), hipMemcpyHostToDevice);

    // 4. 配置 CUDA 核心
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    bit_reversal_reorder_kernel<<<gridSize, blockSize>>>(d_input, d_output, d_indices, n);

    // 5. 將重排結果拷回 CPU
    hipMemcpy(data, d_output, n * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

    // 6. 釋放記憶體
    free(bit_reversed_indices);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_indices);
}

__global__ void fft_kernel(thrust::complex<double>* data, int n, int stage) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int m = 1 << stage;           // Size of subsequences
    int m2 = m >> 1;              // Half of m

    if (idx < n / 2) {
        int group = idx / m2;
        int pair = idx % m2;

        int pos = group * m + pair;

        thrust::complex<double> w = thrust::exp(thrust::complex<double>(0, -2.0 * M_PI * pair / m));

        thrust::complex<double> t = w * data[pos + m2];
        thrust::complex<double> u = data[pos];

        data[pos] = u + t;
        data[pos + m2] = u - t;
    }
}

void gpu_fft(thrust::complex<double>* d_data, int n) {
    int log_n = log2(n);

    for (int stage = 1; stage <= log_n; ++stage) {
        // int m = 1 << stage;
        int threadsPerBlock = 256;
        int numButterflies = n / 2;
        int numBlocks = (numButterflies + threadsPerBlock - 1) / threadsPerBlock;

        fft_kernel<<<numBlocks, threadsPerBlock>>>(d_data, n, stage);

        // Synchronize to ensure all threads have completed
        hipDeviceSynchronize();
    }
}

// FFT function (FFT)
inline void fft(vector<ComplexD>& data) {
    int n = data.size();

    // Copy data to device
    thrust::complex<double>* d_data;
    hipMalloc(&d_data, n * sizeof(thrust::complex<double>));
    hipMemcpy(d_data, data.data(), n * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

    // Perform bit reversal on GPU
    gpu_bit_reversal(d_data, n);

    // Perform FFT on GPU
    gpu_fft(d_data, n);

    // Copy result back to host
    hipMemcpy(data.data(), d_data, n * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_data);
}

// CUDA kernel for frequency centering
__global__ void center_frequency_kernel(ComplexGPU* data, int height, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int half_h = height / 2;
    int half_w = width / 2;
    
    // 只處理左上象限的點，對應的其他三個象限會自動完成交換
    if (idx < half_w && idy < half_h) {
        // 計算四個要交換的位置
        int pos1 = idy * width + idx;                          // 左上
        int pos2 = (idy + half_h) * width + (idx + half_w);   // 右下
        int pos3 = idy * width + (idx + half_w);              // 右上
        int pos4 = (idy + half_h) * width + idx;              // 左下
        
        // 交換數據
        ComplexGPU temp1 = data[pos1];
        ComplexGPU temp2 = data[pos3];
        
        data[pos1] = data[pos2];
        data[pos2] = temp1;
        data[pos3] = data[pos4];
        data[pos4] = temp2;
    }
}

void center_frequency(MatrixD& data, int height, int width) {
    // 分配 GPU 記憶體並複製數據
    ComplexGPU* d_data;
    size_t size = height * width * sizeof(ComplexGPU);
    
    // 將 2D vector 轉換為 1D array
    ComplexGPU* h_data = new ComplexGPU[height * width];
    for(int i = 0; i < height; i++) {
        for(int j = 0; j < width; j++) {
            h_data[i * width + j] = ComplexGPU(data[i][j].real(), data[i][j].imag());
        }
    }
    
    // 分配 GPU 記憶體並複製數據
    hipMalloc(&d_data, size);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);
    
    // 設定 kernel 執行配置
    dim3 blockDim(16, 16);
    dim3 gridDim(
        (width/2 + blockDim.x - 1) / blockDim.x,
        (height/2 + blockDim.y - 1) / blockDim.y
    );
    
    // 執行 kernel
    center_frequency_kernel<<<gridDim, blockDim>>>(d_data, height, width);
    
    // 複製結果回 host
    hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);
    
    // 將結果轉回 MatrixD 格式
    for(int i = 0; i < height; i++) {
        for(int j = 0; j < width; j++) {
            data[i][j] = ComplexD(h_data[i * width + j].real(), h_data[i * width + j].imag());
        }
    }
    
    // 釋放記憶體
    delete[] h_data;
    hipFree(d_data);
    
    // 錯誤檢查
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}

// 修改 fft kernel 以處理特定的一行/列
__global__ void fft_kernel_row(thrust::complex<double>* data, int n, int stage, int row, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int m = 1 << stage;           // Size of subsequences
    int m2 = m >> 1;              // Half of m

    if (idx < n / 2) {
        int group = idx / m2;
        int pair = idx % m2;
        int pos = row * width + group * m + pair;

        thrust::complex<double> w = thrust::exp(thrust::complex<double>(0, -2.0 * M_PI * pair / m));
        thrust::complex<double> t = w * data[pos + m2];
        thrust::complex<double> u = data[pos];

        data[pos] = u + t;
        data[pos + m2] = u - t;
    }
}

__global__ void fft_kernel_col(thrust::complex<double>* data, int n, int stage, int col, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int m = 1 << stage;           // Size of subsequences
    int m2 = m >> 1;              // Half of m

    if (idx < n / 2) {
        int group = idx / m2;
        int pair = idx % m2;
        int pos = (group * m + pair) * width + col;

        thrust::complex<double> w = thrust::exp(thrust::complex<double>(0, -2.0 * M_PI * pair / m));
        thrust::complex<double> t = w * data[pos + m2 * width];
        thrust::complex<double> u = data[pos];

        data[pos] = u + t;
        data[pos + m2 * width] = u - t;
    }
}

// 修改 GPU 上的 bit reversal，使其能夠處理跨步存取
__global__ void bit_reversal_col_kernel(thrust::complex<double>* data, int* indices, int n, int col, int width) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        int reversed_idx = indices[idx];
        if (idx < reversed_idx) {
            thrust::complex<double> temp = data[idx * width + col];
            data[idx * width + col] = data[reversed_idx * width + col];
            data[reversed_idx * width + col] = temp;
        }
    }
}

// 用於計算最大值的 kernel
__global__ void compute_max_val_kernel(thrust::complex<double>* data, double* max_val, int height, int width, int w) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < width && idy < height) {
        double val = log(1 + thrust::abs(data[idy * w + idx]));
        atomicMax((unsigned long long*)max_val, __double_as_longlong(val));
    }
}

// 用於將複數數據轉換為圖像數據的 kernel
__global__ void convert_to_image_kernel(thrust::complex<double>* data, unsigned char* output, 
                                      double* max_val, int height, int width, int w) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < width && idy < height) {
        double val = log(1 + thrust::abs(data[idy * w + idx]));
        output[idy * width + idx] = static_cast<unsigned char>((val / *max_val) * 255);
    }
}

void fft_process_2d(unsigned char* s, unsigned char* t, int height, int width) {
    int w = power_of_two(width);
    int h = power_of_two(height);
    size_t complex_size = h * w * sizeof(thrust::complex<double>);
    
    // 在 GPU 上分配記憶體
    thrust::complex<double> *d_data;
    hipMalloc(&d_data, complex_size);
    
    // 分配並初始化 bit reversal 索引表
    int* h_indices = (int*)malloc(max(w, h) * sizeof(int));
    int* d_indices;
    hipMalloc(&d_indices, max(w, h) * sizeof(int));
    
    // 將輸入數據轉換為複數並複製到 GPU
    vector<thrust::complex<double>> h_data(h * w);
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            h_data[i * w + j] = thrust::complex<double>((double)s[i * width + j], 0);
        }
        // 填充剩餘部分為 0
        for (int j = width; j < w; ++j) {
            h_data[i * w + j] = thrust::complex<double>(0, 0);
        }
    }
    // 填充剩餘行為 0
    for (int i = height; i < h; ++i) {
        for (int j = 0; j < w; ++j) {
            h_data[i * w + j] = thrust::complex<double>(0, 0);
        }
    }
    
    hipMemcpy(d_data, h_data.data(), complex_size, hipMemcpyHostToDevice);
    
    // 對每一行進行 FFT
    int threadsPerBlock = 256;
    int numBlocks = (w / 2 + threadsPerBlock - 1) / threadsPerBlock;
    
    // 先生成行的 bit reversal 索引
    generate_bit_reversal_indices(h_indices, w);
    hipMemcpy(d_indices, h_indices, w * sizeof(int), hipMemcpyHostToDevice);
    
    // 對每一行進行 FFT
    for (int i = 0; i < h; ++i) {
        // 先對該行進行 bit reversal
        thrust::complex<double>* row = d_data + i * w;
        gpu_bit_reversal(row, w);
        
        // 對該行執行 FFT
        for (int stage = 1; stage <= (int)log2(w); ++stage) {
            fft_kernel_row<<<numBlocks, threadsPerBlock>>>(d_data, w, stage, i, w);
            hipDeviceSynchronize();
        }
    }
    
    // 生成列的 bit reversal 索引
    generate_bit_reversal_indices(h_indices, h);
    hipMemcpy(d_indices, h_indices, h * sizeof(int), hipMemcpyHostToDevice);
    
    // 對每一列進行 FFT
    numBlocks = (h + threadsPerBlock - 1) / threadsPerBlock;
    for (int j = 0; j < w; ++j) {
        // 對該列進行 bit reversal
        bit_reversal_col_kernel<<<numBlocks, threadsPerBlock>>>(d_data, d_indices, h, j, w);
        hipDeviceSynchronize();
        
        // 對該列執行 FFT
        for (int stage = 1; stage <= (int)log2(h); ++stage) {
            fft_kernel_col<<<(h/2 + threadsPerBlock - 1) / threadsPerBlock, threadsPerBlock>>>(
                d_data, h, stage, j, w);
            hipDeviceSynchronize();
        }
    }
    
    // 中心化頻率
    dim3 blockDim(16, 16);
    dim3 gridDim((w + blockDim.x - 1) / blockDim.x,
                 (h + blockDim.y - 1) / blockDim.y);
    center_frequency_kernel<<<gridDim, blockDim>>>(d_data, h, w);
    hipDeviceSynchronize();
    
    // 計算頻譜幅度並正規化
    hipMemcpy(h_data.data(), d_data, complex_size, hipMemcpyDeviceToHost);
    
    // 分配設備上的輸出圖像記憶體和最大值記憶體
    unsigned char* d_output;
    double* d_max_val;
    hipMalloc(&d_output, height * width * sizeof(unsigned char));
    hipMalloc(&d_max_val, sizeof(double));
    
    // 初始化最大值為 0
    hipMemset(d_max_val, 0, sizeof(double));
    
    // 計算最大值
    compute_max_val_kernel<<<gridDim, blockDim>>>(d_data, d_max_val, height, width, w);
    hipDeviceSynchronize();
    
    // 轉換為圖像數據
    convert_to_image_kernel<<<gridDim, blockDim>>>(d_data, d_output, d_max_val, height, width, w);
    hipDeviceSynchronize();
    
    // 將結果複製回主機
    hipMemcpy(t, d_output, height * width * sizeof(unsigned char), hipMemcpyDeviceToHost);
    
    // 釋放記憶體
    hipFree(d_output);
    hipFree(d_max_val);
    hipFree(d_data);
    hipFree(d_indices);
    free(h_indices);
}

// CUDA kernel for grayscale conversion
__global__ void convert_to_gray_kernel(unsigned char *s, unsigned char *t, int height, int width, int channels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx < width && idy < height) {
        int pos = idy * width + idx;
        int pos_rgb = idy * width * channels + idx * channels;
        
        int R = s[pos_rgb + 2];
        int G = s[pos_rgb + 1];
        int B = s[pos_rgb + 0];
        
        t[pos] = 0.299f * R + 0.587f * G + 0.114f * B;
    }
}

void convert_to_gray(unsigned char *s, unsigned char *t, int height, int width, int channels) {
    unsigned char *d_src, *d_dst;
    size_t size_src = height * width * channels * sizeof(unsigned char);
    size_t size_dst = height * width * sizeof(unsigned char);
    
    // Allocate device memory
    hipMalloc(&d_src, size_src);
    hipMalloc(&d_dst, size_dst);
    
    // Copy input image to device
    hipMemcpy(d_src, s, size_src, hipMemcpyHostToDevice);
    
    // Configure kernel launch parameters
    dim3 blockDim(32, 32);
    dim3 gridDim(
        (width + blockDim.x - 1) / blockDim.x,
        (height + blockDim.y - 1) / blockDim.y
    );
    
    // Launch kernel
    convert_to_gray_kernel<<<gridDim, blockDim>>>(d_src, d_dst, height, width, channels);
    
    // Copy result back to host
    hipMemcpy(t, d_dst, size_dst, hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_src);
    hipFree(d_dst);
    
    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
}

int main(int argc, char **argv) {
    struct timespec start, end;
    double time_used;
    
    // 開始計時
    clock_gettime(CLOCK_MONOTONIC, &start);

    unsigned height, width, channels;
    unsigned char *src = NULL, *gray, *dst;

    read_png(argv[1], &src, &height, &width, &channels);
    gray = (unsigned char *) malloc(height * width * sizeof(unsigned char));
    dst = (unsigned char *) malloc(height * width * sizeof(unsigned char));

    convert_to_gray(src, gray, height, width, channels);

    fft_process_2d(gray, dst, height, width);

    write_png(argv[2], dst, height, width, 1);

    free(src);
    free(gray);
    free(dst);

    // 結束計時
    clock_gettime(CLOCK_MONOTONIC, &end);
    
    // 計算執行時間（轉換為毫秒）
    time_used = (end.tv_sec - start.tv_sec) * 1000.0 + 
                (end.tv_nsec - start.tv_nsec) / 1000000.0;
    
    printf("Total execution time: %.2f ms\n", time_used);
    return 0;
}